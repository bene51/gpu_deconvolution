#include "hip/hip_runtime.h"
#include "fmvd_deconvolve_cuda.cuh"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "fmvd_cuda_utils.h"

/**
 * Resize the kernel to fftW and fftH, padding it with zeros and
 * positioning it such that its center is at (0, 0).
 */
__global__ void padKernel_kernel(
		float *d_PaddedKernel,
		float *d_Kernel,
		int fftH,
		int fftW,
		int kernelH,
		int kernelW,
		int kernelY,
		int kernelX
		)
{
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	const int x = blockDim.x * blockIdx.x + threadIdx.x;

	if (y < kernelH && x < kernelW) {
		int ky = y - kernelY;
		if (ky < 0)
			ky += fftH;

		int kx = x - kernelX;
		if (kx < 0)
			kx += fftW;

		d_PaddedKernel[ky * fftW + kx] = d_Kernel[y * kernelW + x];
	}
}

extern "C" void padKernel(
		float *d_Dst,
		float *d_Src,
		int fftH,
		int fftW,
		int kernelH,
		int kernelW,
		hipStream_t stream
		)
{
	assert(d_Src != d_Dst);
	dim3 threads(32, 8);
	dim3 grid(iDivUp(kernelW, threads.x), iDivUp(kernelH, threads.y));

	const int kernelY = kernelH / 2;
	const int kernelX = kernelW / 2;

	padKernel_kernel<<<grid, threads, 0, stream>>>(
			d_Dst,
			d_Src,
			fftH,
			fftW,
			kernelH,
			kernelW,
			kernelY,
			kernelX
			);
	getLastCudaError("padKernel_kernel<<<>>> execution failed\n");
}


__global__ void padWeights_kernel(
		float *d_PaddedWeights,
		float *d_PaddedWeightSums,
		float *d_Weights,
		int fftH,
		int fftW,
		int dataH,
		int dataW,
		int kernelH,
		int kernelW,
		int kernelY,
		int kernelX
		)
{
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	const int x = blockDim.x * blockIdx.x + threadIdx.x;

	const int borderH = dataH + kernelY;
	const int borderW = dataW + kernelX;

	if (y < fftH && x < fftW)
	{
		int dy, dx, idx;
		float v;

		if (y < dataH)
			dy = y;

		if (x < dataW)
			dx = x;

		if (y >= dataH && y < borderH)
			dy = dataH - 1;

		if (x >= dataW && x < borderW)
			dx = dataW - 1;

		if (y >= borderH)
			dy = 0;

		if (x >= borderW)
			dx = 0;

		v = d_Weights[dy * dataW + dx];
		idx = y * fftW + x;

		d_PaddedWeights[idx] = v;
		d_PaddedWeightSums[idx] += v;
	}
}

extern "C" void padWeights(
		float *d_PaddedWeights,
		float *d_PaddedWeightSums,
		float *d_Weights,
		int fftH,
		int fftW,
		int dataH,
		int dataW,
		int kernelH,
		int kernelW,
		hipStream_t stream
		)
{
	dim3 threads(32, 8);
	dim3 grid(
			iDivUp(fftW, threads.x),
			iDivUp(fftH, threads.y));

	const int kernelY = kernelH / 2;
	const int kernelX = kernelW / 2;

	padWeights_kernel<<<grid, threads, 0, stream>>>(
			d_PaddedWeights,
			d_PaddedWeightSums,
			d_Weights,
			fftH,
			fftW,
			dataH,
			dataW,
			kernelH,
			kernelW,
			kernelY,
			kernelX
			);
	getLastCudaError("padWeights<<<>>> execution failed\n");
}


__global__ void normalizeWeights_kernel(
		float *d_PaddedWeights,
		float *d_PaddedWeightSums,
		int fftH,
		int fftW
		)
{
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	const int x = blockDim.x * blockIdx.x + threadIdx.x;

	if (y < fftH && x < fftW)
	{
		int idx = y * fftW + x;
		float d = d_PaddedWeightSums[idx];
		if(d > 0)
			d_PaddedWeights[idx] /= d;
	}
}

extern "C" void normalizeWeights(
		float *d_PaddedWeights,
		float *d_PaddedWeightSums,
		int fftH,
		int fftW,
		hipStream_t stream
		)
{
	dim3 threads(32, 8);
	dim3 grid(
			iDivUp(fftW, threads.x),
			iDivUp(fftH, threads.y));

	normalizeWeights_kernel<<<grid, threads, 0, stream>>>(
			d_PaddedWeights,
			d_PaddedWeightSums,
			fftH,
			fftW
			);
	getLastCudaError("normalizeWeights_kernel<<<>>> execution failed\n");
}

__global__ void padDataClampToBorder32_kernel(
		float *d_PaddedData,
		float *d_Data,
		int fftH,
		int fftW,
		int dataH,
		int dataW,
		int kernelH,
		int kernelW,
		int kernelY,
		int kernelX
		)
{
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int borderH = dataH + kernelY;
	const int borderW = dataW + kernelX;

	if (y < fftH && x < fftW) {
		int dy, dx;

		if (y < dataH)
			dy = y;

		if (x < dataW)
			dx = x;

		if (y >= dataH && y < borderH)
			dy = dataH - 1;

		if (x >= dataW && x < borderW)
			dx = dataW - 1;

		if (y >= borderH)
			dy = 0;

		if (x >= borderW)
			dx = 0;

		d_PaddedData[y * fftW + x] = d_Data[dy * dataW + dx];
	}
}

extern "C" void padDataClampToBorder32(
		float *d_PaddedData,
		float *d_Data,
		int fftH,
		int fftW,
		int dataH,
		int dataW,
		int kernelH,
		int kernelW,
		hipStream_t stream
		)
{
	assert(d_PaddedData != d_Data);
	dim3 threads(32, 8);
	dim3 grid(
			iDivUp(fftW, threads.x),
			iDivUp(fftH, threads.y));

	const int kernelY = kernelH / 2;
	const int kernelX = kernelW / 2;

	padDataClampToBorder32_kernel<<<grid, threads, 0, stream>>>(
			d_PaddedData,
			d_Data,
			fftH,
			fftW,
			dataH,
			dataW,
			kernelH,
			kernelW,
			kernelY,
			kernelX
			);
	getLastCudaError("padDataClampToBorder32_kernel<<<>>> execution failed\n");
}


__global__ void unpadData32_kernel(
		float *d_Data,
		float *d_PaddedData,
		int fftH,
		int fftW,
		int dataH,
		int dataW
		)
{
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	const int x = blockDim.x * blockIdx.x + threadIdx.x;

	if (y < dataH && x < dataW)
		d_Data[y * dataW + x] = d_PaddedData[y * fftW + x];
}

extern "C" void unpadData32(
		float *d_Dst,
		float *d_Src,
		int fftH,
		int fftW,
		int dataH,
		int dataW,
		hipStream_t stream
		)
{
	dim3 threads(32, 8);
	dim3 grid(
			iDivUp(dataW, threads.x),
			iDivUp(dataH, threads.y));

	unpadData32_kernel<<<grid, threads, 0, stream>>>(
			d_Dst,
			d_Src,
			fftH,
			fftW,
			dataH,
			dataW
			);
	getLastCudaError("unpadData_kernel<<<>>> execution failed\n");
}

/**
 * Modulate Fourier image of padded data by Fourier image of padded kernel
 * and normalize by FFT size
 */
inline __device__ void mulAndScale(fComplex &a, const fComplex &b, const float &c)
{
	fComplex t = {c *(a.x * b.x - a.y * b.y), c *(a.y * b.x + a.x * b.y)};
	a = t;
}

__global__ void modulateAndNormalize_kernel(
		fComplex *d_Dst,
		fComplex *d_Src,
		int dataSize,
		float c
		)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= dataSize)
		return;

	fComplex a = d_Src[i];
	fComplex b = d_Dst[i];

	mulAndScale(a, b, c);

	d_Dst[i] = a;
}

extern "C" void modulateAndNormalize(
		fComplex *d_Dst,
		fComplex *d_Src,
		int fftH,
		int fftW,
		int padding,
		hipStream_t stream
		)
{
	assert(fftW % 2 == 0);
	const int dataSize = fftH * (fftW / 2 + padding);

	modulateAndNormalize_kernel<<<iDivUp(dataSize, 256), 256, 0, stream>>>(
			d_Dst,
			d_Src,
			dataSize,
			1.0f / (float)(fftW *fftH)
			);
	getLastCudaError("modulateAndNormalize() execution failed\n");
}

__global__ void multiply32_kernel(
		float *d_a,
		float *d_b,
		float *weights,
		float *d_dest,
		int dataSize
		)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= dataSize)
		return;

	float target = d_a[i] * d_b[i];
	float change = target - d_dest[i];
	float weight = weights[i];
	change *= weight;
	d_dest[i] += change;
}

extern "C" void multiply32(
		float *d_a,
		float *d_b,
		float *d_weights,
		float *d_dest,
		int fftH,
		int fftW,
		hipStream_t stream
		)
{
	const int dataSize = fftH * fftW;

	multiply32_kernel<<<iDivUp(dataSize, 256), 256, 0, stream>>>(
			d_a,
			d_b,
			d_weights,
			d_dest,
			dataSize
			);
	getLastCudaError("multiply32_kernel<<<>>> execution failed\n");
}

#define SAMPLE              unsigned short
#define BITS_PER_SAMPLE     16 
#include "fmvd_deconvolve_cuda.impl.cu"

#define SAMPLE              unsigned char
#define BITS_PER_SAMPLE     8
#include "fmvd_deconvolve_cuda.impl.cu"

